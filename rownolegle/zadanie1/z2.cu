#include<stdio.h>
#include<hip/hip_runtime.h>
#include<time.h>
__global__ void zad2(float *a,float *b,float *c, int N)
{
 int idx = blockIdx.x * blockDim.x + threadIdx.x;
 if(idx<N) c[idx] = a[idx] + b[idx];
}

int main(void)
{
clock_t t1,t2,t5,t6,t7,t8;
float *a_h,*b_h,*c_h;
float *a_d,*b_d,*c_d;
const int N = 50000000;

size_t size = N * sizeof(float);
t1=clock();
a_h = (float *)malloc(size);
b_h = (float *)malloc(size);
c_h = (float *)malloc(size);
hipMalloc((void **) &a_d,size);
hipMalloc((void **) &b_d,size);
hipMalloc((void **) &c_d,size);
t2=clock();

for(int i=0;i<N;i++)
{
 a_h[i]=(float)(i+1);
 b_h[i]=(float)(i+1);
 c_h[i]=(float)(i+1);
}
hipEvent_t start, stop;
float time;
hipEventCreate(&start);
hipEventCreate(&stop);

t5=clock();
hipMemcpy(a_d,a_h,size,hipMemcpyHostToDevice);
hipMemcpy(b_d,b_h,size,hipMemcpyHostToDevice);
hipMemcpy(c_d,c_h,size,hipMemcpyHostToDevice);
t6=clock();

int block_size = 1024;
int n_blocks = N/block_size + (N%block_size == 0 ? 0:1);

hipEventRecord(start, 0);
zad2<<<n_blocks,block_size>>>(a_d,b_d,c_d,N);
hipEventRecord(stop, 0);
hipEventSynchronize(stop);

t7=clock();
hipMemcpy(a_h,a_d,size,hipMemcpyDeviceToHost);
hipMemcpy(b_h,b_d,size,hipMemcpyDeviceToHost);
hipMemcpy(c_h,c_d,size,hipMemcpyDeviceToHost);
t8=clock();

//for(int i=0;i<N;i++)
//{
//printf("%d rekord to: %f\n",i,c_h[i]);
//}

printf("Czas alokowania danych: %f s\n",(float)(t2-t1)/CLOCKS_PER_SEC);

printf("Czas przesyłu danych: %f s\n",(float)((t6-t5)+(t8-t7))/CLOCKS_PER_SEC);
free(a_h);
free(b_h);
free(c_h);
hipFree(a_d);
hipFree(b_d);
hipFree(c_d);
hipEventElapsedTime(&time, start, stop);
printf ("Czas wykonania dodawania wektorów: %f ms\n", time);

}


