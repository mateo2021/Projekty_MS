#include<stdio.h>
#include<hip/hip_runtime.h>
#include<time.h>
#include<math.h>

__global__ void zad3(float *a,float *b,float *c, int N)
{
 int idx = blockIdx.x * blockDim.x + threadIdx.x;
 if(idx<N) c[idx] = a[idx] + b[idx];
}

int main(void)
{

clock_t t1,t2;
float *a_h,*b_h,*c_h;
float *a_d,*b_d,*c_d;
const int N = 50000000;
hipEvent_t start1,start2,start3,start4,start5,stop1,stop2,stop3,stop4,stop5;
float time1,time2,time3,time4,time5;

hipEventCreate(&start1);
hipEventCreate(&start2);
hipEventCreate(&stop1);
hipEventCreate(&stop2);
hipEventCreate(&stop3);
hipEventCreate(&stop4);
hipEventCreate(&stop5);
hipEventCreate(&start3);
hipEventCreate(&start4);
hipEventCreate(&start5);

size_t size = N * sizeof(float);

t1=clock();
hipHostMalloc((void**)&a_h,size, hipHostMallocDefault);  //alokowanie na hoscie (pamiec przypinana)
hipHostMalloc((void**)&b_h,size, hipHostMallocDefault);
hipHostMalloc((void**)&c_h,size, hipHostMallocDefault);

hipMalloc((void **) &a_d,size);   //alokowanie pamieci na device 
hipMalloc((void **) &b_d,size);
hipMalloc((void **) &c_d,size);
t2=clock();

for(int i=0;i<N;i++)
{
 a_h[i]=(float)(i+1);
 b_h[i]=(float)(i+1);
 c_h[i]=(float)(i+1);
}

hipStream_t strumien1, strumien2;
hipStreamCreate(&strumien1);
hipStreamCreate(&strumien2);

hipEventRecord(start3, 0);
hipMemcpyAsync(a_d,a_h,size/2,hipMemcpyHostToDevice,strumien1);
hipMemcpyAsync(b_d,b_h,size/2,hipMemcpyHostToDevice,strumien1);
hipMemcpyAsync(c_d,c_h,size/2,hipMemcpyHostToDevice,strumien1);
hipEventRecord(stop3, 0);
hipEventSynchronize(stop3);

hipEventRecord(start1, 0);
 zad3<<<N/2/N+1,N,0,strumien1>>>(a_d,b_d,c_d,N/2);
hipEventRecord(stop1, 0);
hipEventSynchronize(stop1);

hipEventRecord(start4, 0);
hipMemcpyAsync(a_h,a_d,size/2,hipMemcpyDeviceToHost,strumien1); 
hipMemcpyAsync(b_h,b_d,size/2,hipMemcpyDeviceToHost,strumien1); 
hipMemcpyAsync(c_h,c_d,size/2,hipMemcpyDeviceToHost,strumien1);
hipMemcpyAsync(a_d+N/2,a_h+N/2,size/2,hipMemcpyHostToDevice,strumien2);
hipMemcpyAsync(b_d+N/2,b_h+N/2,size/2,hipMemcpyHostToDevice,strumien2);
hipMemcpyAsync(c_d+N/2,c_h+N/2,size/2,hipMemcpyHostToDevice,strumien2);
hipEventRecord(stop4, 0);
hipEventSynchronize(stop4);

hipEventRecord(start2, 0);
zad3<<<N/2/N+1,N,0,strumien2>>>(a_d+N/2,b_d+N/2,c_d+N/2,N/2);
hipEventRecord(stop2, 0);
hipEventSynchronize(stop2);

hipEventRecord(start5, 0);
hipMemcpyAsync(a_h+N/2,a_d+N/2,size/2,hipMemcpyDeviceToHost,strumien2);
hipMemcpyAsync(b_h+N/2,b_d+N/2,size/2,hipMemcpyDeviceToHost,strumien2);
hipMemcpyAsync(c_h+N/2,c_d+N/2,size/2,hipMemcpyDeviceToHost,strumien2);
hipEventRecord(stop5, 0);
hipEventSynchronize(stop5);

printf("Czas alokowania danych: %f s\n",(float)(t2-t1)/CLOCKS_PER_SEC);
//printf("Czas przesyłu danych: %f s\n",(float)(((t4-t3)+(t6-t5)+(t8-t7)))/CLOCKS_PER_SEC);

hipStreamDestroy(strumien1);
hipStreamDestroy(strumien2);
hipHostFree(a_h);
hipHostFree(b_h);
hipHostFree(c_h);
hipFree(a_d);
hipFree(b_d);
hipFree(c_d);

hipEventElapsedTime(&time1, start1, stop1);
hipEventElapsedTime(&time2, start2, stop2);
hipEventElapsedTime(&time3, start3, stop3);
hipEventElapsedTime(&time4, start4, stop4);
hipEventElapsedTime(&time5, start5, stop5);

printf ("Czas wykonania dodawania wektorów: %f ms\n", time1+time2);
printf("Czas przesyłu danych: %f ms\n",time3+time4+time5);

}


