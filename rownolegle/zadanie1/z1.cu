#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>



int main(void) {

int *a_h;
int *b_h;
int *c_h;
int N = 50000000;
size_t size = N * sizeof(int);

hipEvent_t start,stop,start1,stop1;
float time,time1;
hipEventCreate(&start);
hipEventCreate(&stop);
hipEventCreate(&start1);
hipEventCreate(&stop1);


hipEventRecord(start, 0);
 a_h = (int *)malloc(size);
 b_h = (int *)malloc(size);
 c_h = (int *)malloc(size);
hipEventRecord(stop, 0);
hipEventSynchronize(stop);


 for(int i = 0; i < N; i++){ 
	a_h[i] = i;
        b_h[i]= i;
        c_h[i]=i;
	}
hipEventRecord(start1, 0);
 for(int i = 0; i < N;i++){
         c_h[i]=a_h[i]+b_h[i];
  }
hipEventRecord(stop1, 0);
hipEventSynchronize(stop1);


//printf("czas alokowania %f \n",(float)(t2-t1)/CLOCKS_PER_SEC);
//printf("czas wykonywania programu %f \n",(float)(t4-t3)/CLOCKS_PER_SEC);
 free(a_h);
 free(b_h);
 free(c_h);

hipEventElapsedTime(&time1, start1, stop1);
 printf("Czas wykonania programu: %f ms\n",time1);
hipEventElapsedTime(&time, start, stop);
 printf("Czas alokowania: %f ms\n",time);
 return 0;
}
